#include "hip/hip_runtime.h"
// ****************************************************************************
//
//	rx_slicescreen_fms_1x3.c
//
// ****************************************************************************
//
//	Copyright 2017 by Radex AG, Switzerland. All rights reserved.
//	Written by Gilbert Therville 
//
// ****************************************************************************

//--- includes ---------------------------------------------------------------------
#include "hip/hip_runtime.h"
#include ""

#ifdef __cplusplus
extern "C"{
#endif

#include "rx_def.h"
#include "rx_error.h"
#include "rx_trace.h"
#include "rx_threads.h"
#include "rx_mem.h"
#include "rx_slicescreen.h"
#include "gpu.h"

#ifdef linux
	#include "errno.h"
#endif

//--- defines ----------------------------------------------------------
#define TA_WIDTH	256
#define TA_HEIGHT	256

//	time per color and scan of "Tiger-Woman" image (~full width) on "GeForce GT 1030"
//	int		sectors=1;		// 300 ms
//	int		sectors=4;		// 130 ms
//	int		sectors=8;		// 83 ms
//	int		sectors=16;		// 60 ms
#define	SECTORS 32			// 30 ms

#define STREAM_CNT	4

typedef struct
{
	hipStream_t	stream;

	UINT32			in_size;
	BYTE*			in;

	UINT32			out_size;
	BYTE*			out;

	UINT16*			ta;
	UINT16*			df;	// density_factor

	int				time[4];
	int				Time[3];
} SStreamPar;

//--- static variables -------------------------------------------------
static int				_Init=FALSE;

static hipDeviceProp_t	_GpuProp;
static int				_GPU_Present=-1;

static int				_Time[3];

static SStreamPar		_GPU_Stream[STREAM_CNT];

//--- prototypes ----------------------------------------------------
static int _gpu_malloc(SSLiceInfo *inplane, SSLiceInfo *outplane, int bitsPerPixel, SStreamPar *par);

//--- gpu_is_board_present -----------------------------
int gpu_is_board_present(void)
{
	if (_GPU_Present<0)
	{
		hipGetDeviceProperties(&_GpuProp, 0);
		if (_GpuProp.name[0]) TrPrintf(TRUE, "GPU: %s", _GpuProp.name);
		else				  TrPrintf(TRUE, "GPU: not present");	
		_GPU_Present = (_GpuProp.name[0]!=0);

		#ifdef DEBUG
			if (FALSE && _GPU_Present)
		//	if (_GPU_Present)
			{
				_GPU_Present = FALSE;
				Error(WARN, 0, "Disable GPU for tests");
			}
		#endif
	}
	return _GPU_Present;
}

//--- gpu_init -------------------------------
int gpu_init(void)
{
	if (!_Init)
	{
		_Init = TRUE;
		memset(_GPU_Stream, 0, sizeof(_GPU_Stream));
		_GPU_Present = gpu_is_board_present();
	}
	Error(LOG, 0, "Screening: GPU=>>%s<<", _GpuProp.name);
	if (!_GPU_Present && rx_def_is_tx((EPrinterType)RX_Spooler.printerType)) Error(WARN, 0, "GPU not present!");

	if (_GPU_Present) return REPLY_OK;
	else return REPLY_ERROR;
}

//--- gpu_stream_cnt -------------------
int gpu_stream_cnt(void)
{
	return STREAM_CNT;
}

//--- _gpu_malloc -----------------------------------------------------------
static int _gpu_malloc(SSLiceInfo *inplane, SSLiceInfo *outplane, int bitsPerPixel, SStreamPar *par)
{
	int ret=REPLY_OK;

	if (_GPU_Present)
	{
		if (par->stream==NULL) 
		{
			ret=hipStreamCreate(&par->stream);
			if (ret!=hipSuccess) Error(ERR_ABORT, 0, "could not create CUDA stream, hipError_t=%d", ret);
		}

		if (inplane->dataSize>par->in_size)
		{
			par->in_size = (UINT32)inplane->dataSize;
			if (par->in) ret=hipFree(par->in);
			ret=hipMalloc(&par->in, par->in_size);
		}

		if (outplane->dataSize>par->out_size)
		{
			par->out_size = (UINT32)outplane->dataSize;
			if (par->out) ret=hipFree(par->out);
			ret=hipMalloc(&par->out, par->out_size);
		}

		if (!par->ta) hipMalloc(&par->ta, TA_WIDTH*TA_HEIGHT*sizeof(UINT16));
		if (!par->df) hipMalloc(&par->df, JET_CNT*TA_HEIGHT*sizeof(UINT16));
	}
	return ret;
}

// creates 2 bit/pixel
// if limitL == 0: output = (s,m) else output = (s,m,l)
// need 32 bits for compensating disabled jets
// note that density correction is only used to decide if we jet
// but not to set the drop size

//--- _screen_fms_kernel ----------------------
__global__ void _screen_fms_kernel(UINT8 *in, UINT8 *out, UINT16 *pta, UINT16 *densityFactor, UINT32 height, UINT32 inWidthPx, UINT32 inLineLen, UINT32 outLineLen, UINT32 sectorWidth, UINT32 limitM, UINT32 limitL)
{
	int y = (blockDim.x * blockIdx.x + threadIdx.x)*sectorWidth;
	int x;
	UINT8 *pSrc=&in[y];
	x = y%inLineLen;
	y = y/inLineLen;
	UINT8 *pDst=&out[y*outLineLen+x/4];
	UINT16 *taLine=&pta[(y%TA_HEIGHT)*TA_WIDTH];
	UINT8  dst=0;
	UINT32 limitML = (256 - limitL);
	UINT32 limitSM = ((limitL ? limitL : 256) - limitM);

	UINT32 src;	// need 32 bits for compensating disabled jets
	UINT16  ta;

	sectorWidth+=x;
	if (sectorWidth>inWidthPx) sectorWidth=inWidthPx;

	if (y<height)
	{
		while (x<sectorWidth)
		{
			src = *pSrc++ * densityFactor[x];
			ta  = taLine[x%TA_WIDTH];
			dst <<= 2;
			if (src > ta)
			{
				if (limitL && *pSrc > limitL)
					dst |= (*pSrc - limitL >= limitML * ta / 65536) ? 0x03 : 0x02;
				else if (*pSrc > limitM)
					dst |= (*pSrc - limitM >= limitSM * ta / 65536) ? 0x02 : 0x01;
				else
					dst |= 0x01;
			}

			if (!(++x & 3)) *pDst++=dst;
		}
	}
}

//--- _screen_fms ----------------------
static void _screen_fms(UINT32 y, UINT8 *in, UINT8 *out, UINT16 *pta, UINT16 *densityFactor, UINT32 height, UINT32 inWidthPx, UINT32 inLineLen, UINT32 outLineLen, UINT32 sectorWidth, UINT32 limitM, UINT32 limitL)
{
	UINT32 x;
	UINT8 *pSrc=&in[y];
	x = y%inLineLen;
	y = y/inLineLen;
	UINT8 *pDst=&out[y*outLineLen+x/4];
	UINT16 *taLine=&pta[(y%TA_HEIGHT)*TA_WIDTH];
	UINT8  dst=0;
	UINT32 limitML = (256 - limitL);
	UINT32 limitSM = ((limitL ? limitL : 256) - limitM);

	UINT32 src;	// need 32 bits for compensating disabled jets
	UINT16 ta;

	sectorWidth+=x;
	if (sectorWidth>inWidthPx) sectorWidth=inWidthPx;

	if (y<height)
	{
		while (x<sectorWidth)
		{
			src = *pSrc++ * densityFactor[x];
			ta  = taLine[x%TA_WIDTH];
			dst <<= 2;
			if (src > ta)
			{
				if (limitL && *pSrc > limitL)
					dst |= (*pSrc - limitL >= limitML * ta / 65536) ? 0x03 : 0x02;
				else if (*pSrc > limitM)
					dst |= (*pSrc - limitM >= limitSM * ta / 65536) ? 0x02 : 0x01;
				else
					dst |= 0x01;
			}

			if (!(++x & 3)) *pDst++=dst;
		}
	}
}

//--- _screen_fms_600_kernel ----------------------
__global__ void _screen_fms_600_kernel(UINT8 *in, UINT8 *out, UINT16 *pta, UINT16 *densityFactor, UINT32 height, UINT32 inWidthPx, UINT32 inLineLen, UINT32 outLineLen, UINT32 sectorWidth, UINT32 limitM, UINT32 limitL)
{
	int y = (blockDim.x * blockIdx.x + threadIdx.x)*sectorWidth;
	int x;
	x = 2*(y%inLineLen);
	y = y/inLineLen;
	UINT8 *pSrc=&in[(y/2)*inLineLen+x/2];
	UINT8 *pDst=&out[y*outLineLen+x/4];
	UINT16* taLine=&pta[(y%TA_HEIGHT)*TA_WIDTH];
	UINT8  dst = 0;
	UINT32 limitML = (256 - limitL);
	UINT32 limitSM = ((limitL ? limitL : 256) - limitM);

	UINT32 src;	// need 32 bits for compensating disabled jets
	UINT16  ta;

	sectorWidth=2*sectorWidth+x;
	if (sectorWidth>2*inWidthPx) sectorWidth=2*inWidthPx;

	if (y<2*height)
	{
		while (x<sectorWidth)
		{
			src = *pSrc * densityFactor[x];
			if (x&1) pSrc++;
			ta  = taLine[x%TA_WIDTH];
			dst <<= 2;
			if (src > ta)
			{
				if (limitL && *pSrc > limitL)
					dst |= (*pSrc - limitL >= limitML * ta / 65536) ? 0x03 : 0x02;
				else if (*pSrc > limitM)
					dst |= (*pSrc - limitM >= limitSM * ta / 65536) ? 0x02 : 0x01;
				else
					dst |= 0x01;
			}

			if (!(++x & 3)) *pDst++=dst;
		}
	}
}

//--- _screen_fms_600 ----------------------
static void _screen_fms_600(UINT32 y, UINT8 *in, UINT8 *out, UINT16 *pta, UINT16 *densityFactor, UINT32 height, UINT32 inWidthPx, UINT32 inLineLen, UINT32 outLineLen, UINT32 sectorWidth, UINT32 limitM, UINT32 limitL)
{
	UINT32 x;
	x = y%inLineLen;
	y = y/inLineLen;
	UINT8 *pSrc=&in[(y/2)*inLineLen+x/2];
	UINT8 *pDst=&out[y*outLineLen+x/4];
	UINT16* taLine=&pta[(y%TA_HEIGHT)*TA_WIDTH];
	UINT8  dst=0;
	UINT32 limitML = (256 - limitL);
	UINT32 limitSM = ((limitL ? limitL : 256) - limitM);

	UINT32 src;	// need 32 bits for compensating disabled jets
	UINT16  ta;

	sectorWidth = 2 * sectorWidth + x;
	if (sectorWidth > 2 * inWidthPx) sectorWidth = 2 * inWidthPx;

	if (y<2*height)
	{
		while (x<sectorWidth)
		{
			src = *pSrc * densityFactor[x];
			if (x&1) pSrc++;
			ta  = taLine[x%TA_WIDTH];
			dst <<= 2;
			if (src > ta)
			{
				if (limitL && *pSrc > limitL)
					dst |= (*pSrc - limitL >= limitML * ta / 65536) ? 0x03 : 0x02;
				else if (*pSrc > limitM)
					dst |= (*pSrc - limitM >= limitSM * ta / 65536) ? 0x02 : 0x01;
				else
					dst |= 0x01;
			}

			if (!(++x & 3)) *pDst++=dst;
		}
	}
}

//--- _screen_fms_300_kernel ----------------------
__global__ void _screen_fms_300_kernel(UINT8 *in, UINT8 *out, UINT16 *pta, UINT16 *densityFactor, UINT32 height, UINT32 inWidthPx, UINT32 inLineLen, UINT32 outLineLen, UINT32 sectorWidth, UINT32 limitM, UINT32 limitL)
{
	int y = (blockDim.x * blockIdx.x + threadIdx.x)*sectorWidth;
	int x;
	x = 4*(y%inLineLen);
	y = y/inLineLen;
	UINT8 *pSrc=&in[(y/4)*inLineLen+x/4];
	UINT8 *pDst=&out[y*outLineLen+x/4];
	UINT16* taLine=&pta[(y%TA_HEIGHT)*TA_WIDTH];
	UINT8  dst = 0;
	UINT32 limitML = (256 - limitL);
	UINT32 limitSM = ((limitL ? limitL : 256) - limitM);

	UINT32 src;	// need 32 bits for compensating disabled jets
	UINT16  ta;

	sectorWidth=4*sectorWidth+x;
	if (sectorWidth>4*inWidthPx) sectorWidth=4*inWidthPx;

	if (y<4*height)
	{
		while (x<sectorWidth)
		{
			src = *pSrc * densityFactor[x];
			ta  = taLine[x%TA_WIDTH];
			dst <<= 2;
			if (src > ta)
			{
				if (limitL && *pSrc > limitL)
					dst |= (*pSrc - limitL >= limitML * ta / 65536) ? 0x03 : 0x02;
				else if (*pSrc > limitM)
					dst |= (*pSrc - limitM >= limitSM * ta / 65536) ? 0x02 : 0x01;
				else
					dst |= 0x01;
			}

			if (!(++x & 3)) 
			{
				*pDst++=dst;
				pSrc++;
			}
		}
	}
}

//--- _screen_fms_300 ----------------------
//		300 dpi
static void _screen_fms_300(UINT32 y, UINT8 *in, UINT8 *out, UINT16 *pta, UINT16 *densityFactor, UINT32 height, UINT32 inWidthPx, UINT32 inLineLen, UINT32 outLineLen, UINT32 sectorWidth, UINT32 limitM, UINT32 limitL)
{
	UINT32 x;
	x = 4*(y%inLineLen);
	y = y/inLineLen;
	UINT8 *pSrc=&in[(y/4)*inLineLen+x/4];
	UINT8 *pDst=&out[y*outLineLen+x/4];
	UINT16* taLine=&pta[(y%TA_HEIGHT)*TA_WIDTH];
	UINT8  dst=0;
	UINT32 limitML = (256 - limitL);
	UINT32 limitSM = ((limitL ? limitL : 256) - limitM);
	UINT32 src;	// need 32 bits for compensating disabled jets
	UINT16  ta;

	sectorWidth=4*sectorWidth+x;
	if (sectorWidth>4*inWidthPx) sectorWidth=4*inWidthPx;

	if (y<4*height)
	{
		while (x<sectorWidth)
		{
			src = *pSrc * densityFactor[x];
			ta  = taLine[x%TA_WIDTH];
			dst <<= 2;
			if (src > ta)
			{
				if (limitL && *pSrc > limitL)
					dst |= (*pSrc - limitL >= limitML * ta / 65536) ? 0x03 : 0x02;
				else if (*pSrc > limitM)
					dst |= (*pSrc - limitM >= limitSM * ta / 65536) ? 0x02 : 0x01;
				else
					dst |= 0x01;
			}

			if (!(++x & 3)) 
			{
				*pDst++=dst;
				pSrc++;
			}
		}
	}
}

//--- gpu_blk_size ---------------------------
int gpu_blk_size(void)
{
	return MAX_DENSITY_FACTORS / SECTORS;
}

//--- gpu_screen_FMS_1x3g ----------------------------------------------------------------------
int gpu_screen_FMS_1x3g(SSLiceInfo *inplane, SSLiceInfo *outplane, void *epplaneScreenConfig, const char *dots, int threadNo)
{
	SPlaneScreenConfig* pplaneScreenConfig = (SPlaneScreenConfig*)epplaneScreenConfig;
	UINT16 limitM, limitL;

	SStreamPar	*pstream = &_GPU_Stream[threadNo];

	limitM = 256 * pplaneScreenConfig->limit[0] / 100;
	limitL = 256 * pplaneScreenConfig->limit[1] / 100;
	
	if (pplaneScreenConfig->TA->width!=TA_WIDTH || pplaneScreenConfig->TA->heigth!=TA_HEIGHT) return Error(ERR_ABORT, 0, "TA-ARRAY must be 256*256");

	if (_GPU_Present)
	{
	//	TrPrintfL(TRUE, "GPU[%d].malloc start", threadNo);
		_gpu_malloc(inplane, outplane, pplaneScreenConfig->outputbitsPerPixel, pstream);
	//	TrPrintfL(TRUE, "GPU[%d].malloc done", threadNo);
		pstream->time[0]=rx_get_ticks();
		hipMemcpyAsync(pstream->in, inplane->buffer,				   inplane->lengthPx * inplane->lineLen,		hipMemcpyHostToDevice, pstream->stream);
		hipMemcpyAsync(pstream->ta, pplaneScreenConfig->TA->ta16,	   TA_WIDTH*TA_HEIGHT*sizeof(UINT16),			hipMemcpyHostToDevice, pstream->stream);
		hipMemcpyAsync(pstream->df, pplaneScreenConfig->densityFactor, sizeof(pplaneScreenConfig->densityFactor),	hipMemcpyHostToDevice, pstream->stream);
		hipStreamSynchronize(pstream->stream);
		pstream->time[1]=rx_get_ticks();
	//	TrPrintfL(TRUE, "GPU[%d].memcpy(in) done", threadNo);
		hipMemset(pstream->out, 0x00, outplane->lengthPx * outplane->lineLen);

		// Each Grid consists of X * Y blocks that each contain 512 * 512 threads. Unused threads will just do nothing. (check inside kernel, see code above)
		int sectorWidth		= gpu_blk_size()*inplane->resol.x/DPI_X;
		int threadsPerBlock = 32; //_GpuProp.maxThreadsPerBlock;
		int blocksPerGrid   = (((outplane->lengthPx*inplane->lineLen)/sectorWidth) + threadsPerBlock - 1) / threadsPerBlock;
		if (inplane->resol.x==300)
			_screen_fms_300_kernel <<<blocksPerGrid, threadsPerBlock, 0, pstream->stream >>> (pstream->in, pstream->out, pstream->ta, pstream->df, inplane->lengthPx, inplane->widthPx, inplane->lineLen, outplane->lineLen, sectorWidth, limitM, limitL);
		else if (inplane->resol.x==600)
			_screen_fms_600_kernel <<<blocksPerGrid, threadsPerBlock, 0, pstream->stream >>> (pstream->in, pstream->out, pstream->ta, pstream->df, inplane->lengthPx, inplane->widthPx, inplane->lineLen, outplane->lineLen, sectorWidth, limitM, limitL);
		else
			_screen_fms_kernel <<<blocksPerGrid, threadsPerBlock, 0, pstream->stream >>> (pstream->in, pstream->out, pstream->ta, pstream->df, inplane->lengthPx, inplane->widthPx, inplane->lineLen, outplane->lineLen, sectorWidth, limitM, limitL);

		hipStreamSynchronize(pstream->stream);
		pstream->time[2]=rx_get_ticks();
	//	TrPrintfL(TRUE, "GPU[%d].process done", threadNo);

		hipMemcpyAsync(outplane->buffer, pstream->out, outplane->lengthPx * outplane->lineLen, hipMemcpyDeviceToHost, pstream->stream);

		hipStreamSynchronize(pstream->stream);
		pstream->time[3]=rx_get_ticks();
	//	TrPrintfL(TRUE, "GPU[%d].memcpy(out) done", threadNo);

		pstream->Time[0] = pstream->time[1]-pstream->time[0];
		pstream->Time[1] = pstream->time[2]-pstream->time[1];
		pstream->Time[2] = pstream->time[3]-pstream->time[2];
	}
	else
	{
		int time0=rx_get_ticks();
		int sectorWidth = inplane->widthPx;
		if (inplane->resol.x==300)
		{
			for (UINT32 y=0; y<outplane->lengthPx; y++)
			{
				_screen_fms_300(y*inplane->lineLen, inplane->buffer, outplane->buffer, pplaneScreenConfig->TA->ta16, pplaneScreenConfig->densityFactor, inplane->lengthPx, inplane->widthPx, inplane->lineLen, outplane->lineLen, sectorWidth, limitM, limitL);
			}
		}
		else if (inplane->resol.x==600)
		{
			for (UINT32 y=0; y<outplane->lengthPx; y++)
			{
				_screen_fms_600(y*inplane->lineLen, inplane->buffer, outplane->buffer, pplaneScreenConfig->TA->ta16, pplaneScreenConfig->densityFactor, inplane->lengthPx, inplane->widthPx, inplane->lineLen, outplane->lineLen, sectorWidth, limitM, limitL);
			}
		}
		else
		{
			for (UINT32 y=0; y<outplane->lengthPx; y++)
			{
				_screen_fms(y*inplane->lineLen, inplane->buffer, outplane->buffer, pplaneScreenConfig->TA->ta16, pplaneScreenConfig->densityFactor, inplane->lengthPx, inplane->widthPx, inplane->lineLen, outplane->lineLen, sectorWidth, limitM, limitL);
			}
		}
		_Time[0]=0;
		_Time[1]=rx_get_ticks()-time0;
		_Time[2]=0;
	}
	return REPLY_OK;
}

//--- gpu_time ----------------------
int  gpu_time(int no)
{
	if (_GPU_Present) return (int)(_GPU_Stream[0].Time[no]+0.5);
	else			  return (int)(_Time[no]+0.5);
}

#ifdef __cplusplus
}
#endif