#include "hip/hip_runtime.h"
// ****************************************************************************
//
//	rx_slicescreen_fms_1x3.c
//
// ****************************************************************************
//
//	Copyright 2017 by Radex AG, Switzerland. All rights reserved.
//	Written by Gilbert Therville 
//
// ****************************************************************************

//--- includes ---------------------------------------------------------------------
#include "hip/hip_runtime.h"
#include ""

#ifdef __cplusplus
extern "C"{
#endif

#include "rx_error.h"
#include "rx_trace.h"
#include "rx_threads.h"
#include "rx_mem.h"
#include "rx_slicescreen.h"
#include "rx_slicescreen_fms_1x3_gpu.h"

#ifdef linux
	#include "errno.h"
#endif

//--- defines ----------------------------------------------------------
#define TA_WIDTH	256
#define TA_HEIGHT	256

//	time per color and scan of "Tiger-Woman" image (~full width) on "GeForce GT 1030"
//	int		sectors=1;		// 300 ms
//	int		sectors=4;		// 130 ms
//	int		sectors=8;		// 83 ms
//	int		sectors=16;		// 60 ms
#define	SECTORS 32			// 30 ms

#define STREAM_CNT	4

typedef struct
{
	hipStream_t	stream;

	UINT32			in_size;
	BYTE*			in;

	UINT32			out_size;
	BYTE*			out;

	UINT16*			ta;
	UINT16*			df;	// density_factor

	int				time[4];
	int				Time[3];
} SStreamPar;

//--- static variables -------------------------------------------------
static int		_Init=FALSE;

static hipDeviceProp_t	_GpuProp;
static int				_GPU_Present=-1;

static int				_Time[3];

static SStreamPar _GPU_Stream[STREAM_CNT];

//--- prototypes ----------------------------------------------------
static int _gpu_malloc(SSLiceInfo *inplane, SSLiceInfo *outplane, int bitsPerPixel, SStreamPar *par);

//--- gpu_is_board_present -----------------------------
int gpu_is_board_present(void)
{
	if (_GPU_Present<0)
	{
		hipGetDeviceProperties(&_GpuProp, 0);
		if (_GpuProp.name[0]) TrPrintf(TRUE, "GPU: %s", _GpuProp.name);
		else				  TrPrintf(TRUE, "GPU: not present");	
		_GPU_Present = (_GpuProp.name[0]!=0);
	}
	return _GPU_Present;
}

//--- gpu_init -------------------------------
int gpu_init(void)
{
	if (!_Init)
	{
		_Init = TRUE;
		memset(_GPU_Stream, 0, sizeof(_GPU_Stream));
		_GPU_Present = gpu_is_board_present();
		if (FALSE && _GPU_Present)
		{
			_GPU_Present = FALSE;
			Error(WARN, 0, "Disable GPU for tests");
		}
	}
	Error(LOG, 0, "Screening: GPU=>>%s<<", _GpuProp.name);

	if (_GPU_Present) return REPLY_OK;
	else return REPLY_ERROR;
}

//--- gpu_stream_cnt -------------------
int gpu_stream_cnt(void)
{
	return STREAM_CNT;
}

//--- _gpu_malloc -----------------------------------------------------------
static int _gpu_malloc(SSLiceInfo *inplane, SSLiceInfo *outplane, int bitsPerPixel, SStreamPar *par)
{
	int ret=REPLY_OK;

	if (_GPU_Present)
	{
		if (par->stream==NULL) 
		{
			ret=hipStreamCreate(&par->stream);
			if (ret!=hipSuccess) Error(ERR_ABORT, 0, "could not create CUDA stream, hipError_t=%d", ret);
		}

		if (inplane->dataSize>par->in_size)
		{
			par->in_size = (UINT32)inplane->dataSize;
			if (par->in) ret=hipFree(par->in);
			ret=hipMalloc(&par->in, par->in_size);
		}

		if (outplane->dataSize>par->out_size)
		{
			par->out_size = (UINT32)outplane->dataSize;
			if (par->out) ret=hipFree(par->out);
			ret=hipMalloc(&par->out, par->out_size);
		}

		if (!par->ta) hipMalloc(&par->ta, TA_WIDTH*TA_HEIGHT*sizeof(UINT16));
		if (!par->df) hipMalloc(&par->df, JET_CNT*TA_HEIGHT*sizeof(UINT16));
	}
	return ret;
}

//--- _screen_fms_sml_kernel ----------------------
__global__ void _screen_fms_sml_kernel(UINT8 *in, UINT8 *out, UINT16 *pta, UINT16 *densityFactor, UINT32 lineLen, UINT32 height, UINT32 sectorWidth, UINT32 limitM, UINT32 limitL)
{
	// creates 2 bit/pixel
	// if limitL == 0: output = (s,m) else output = (s,m,l)

	int y = (blockDim.x * blockIdx.x + threadIdx.x)*sectorWidth;
	int x;
	UINT8 *pSrc=&in[y];
	UINT8 *pDst=&out[y/4];
	x = y%lineLen;
	y = y/lineLen;
	UINT16* taLine=&pta[(y%TA_HEIGHT)*TA_WIDTH];
	UINT8  dst;
	UINT32 src;	// need 32 bits for compensating disabled jets
	UINT16  ta;

	if (x+sectorWidth<lineLen) sectorWidth=x+sectorWidth;
	else                       sectorWidth=lineLen;

	if (y<height)
	{
		while (x<sectorWidth)
		{
			src = *pSrc++ * densityFactor[x];
			ta  = taLine[x%TA_WIDTH];
			dst <<= 2;
			if (limitL && src > limitL + ta)	dst |= 0x03;
			else if (src > limitM + ta)			dst |= 0x02;
			else if (src > ta)					dst |= 0x01;

			if (!(++x & 3)) *pDst++=dst;
		}
	}
}

//--- _screen_fms_sml ----------------------
static void _screen_fms_sml(UINT32 y, UINT8 *in, UINT8 *out, UINT16 *pta, UINT16 *densityFactor, UINT32 lineLen, UINT32 height, UINT32 sectorWidth, UINT32 limitM, UINT32 limitL)
{
	// creates 2 bit/pixel
	// if limitL == 0: output = (s,m) else output = (s,m,l)

	UINT32 x;
	UINT8 *pSrc=&in[y];
	UINT8 *pDst=&out[y/4];
	x = y%lineLen;
	y = y/lineLen;
	UINT16* taLine=&pta[(y%TA_HEIGHT)*TA_WIDTH];
	UINT8  dst=0;
	UINT32 src;	// need 32 bits for compensating disabled jets
	UINT16  ta;

	if (x+sectorWidth<lineLen) sectorWidth=x+sectorWidth;
	else					   sectorWidth=lineLen;

	if (y<height)
	{
		while (x<sectorWidth)
		{
			src = *pSrc++ * densityFactor[x];
			ta  = taLine[x%TA_WIDTH];
			dst <<= 2;
			if (limitL && src > limitL + ta)	dst |= 0x03;
			else if (src > limitM + ta)			dst |= 0x02;
			else if (src > ta)					dst |= 0x01;

			if (!(++x & 3)) *pDst++=dst;
		}
	}
}

//--- gpu_blk_size ---------------------------
int gpu_blk_size(void)
{
	return MAX_DENSITY_FACTORS / SECTORS;
}

//--- gpu_screen_FMS_1x3g ----------------------------------------------------------------------
int gpu_screen_FMS_1x3g(SSLiceInfo *inplane, SSLiceInfo *outplane, void *epplaneScreenConfig, const char *dots, int threadNo)
{
	SPlaneScreenConfig* pplaneScreenConfig = (SPlaneScreenConfig*)epplaneScreenConfig;
	UINT16 limitM, limitL;

	SStreamPar	*pstream = &_GPU_Stream[threadNo];

	if (strchr(dots, 'L'))
	{
		limitM = 0x10000 * 33 / 100;
		limitL = 0x10000 * 66 / 100;
	}
	else
	{
		limitM = 0x10000 * 50 / 100;
		limitL = 0;
	}

	if (pplaneScreenConfig->limit[0]) limitM = 65536 * pplaneScreenConfig->limit[0] / 100;
	if (pplaneScreenConfig->limit[1]) limitL = 65536 * pplaneScreenConfig->limit[1] / 100;
	
	if (pplaneScreenConfig->TA->width!=TA_WIDTH || pplaneScreenConfig->TA->heigth!=TA_HEIGHT) return Error(ERR_ABORT, 0, "TA-ARRAY must be 256*256");

	if (_GPU_Present)
	{
	//	TrPrintfL(TRUE, "GPU[%d].malloc start", threadNo);
		_gpu_malloc(inplane, outplane, pplaneScreenConfig->outputbitsPerPixel, pstream);
	//	TrPrintfL(TRUE, "GPU[%d].malloc done", threadNo);
		pstream->time[0]=rx_get_ticks();
		hipMemcpyAsync(pstream->in, inplane->buffer,				   inplane->lengthPx * inplane->lineLen,		hipMemcpyHostToDevice, pstream->stream);
		hipMemcpyAsync(pstream->ta, pplaneScreenConfig->TA->ta16,	   TA_WIDTH*TA_HEIGHT*sizeof(UINT16),			hipMemcpyHostToDevice, pstream->stream);
		hipMemcpyAsync(pstream->df, pplaneScreenConfig->densityFactor, sizeof(pplaneScreenConfig->densityFactor),	hipMemcpyHostToDevice, pstream->stream);
		hipStreamSynchronize(pstream->stream);
		pstream->time[1]=rx_get_ticks();
	//	TrPrintfL(TRUE, "GPU[%d].memcpy(in) done", threadNo);
//		hipMemset(_gpu_out, 0x01, outplane->lengthPx * outplane->lineLen);

		// Each Grid consists of X * Y blocks that each contain 512 * 512 threads. Unused threads will just do nothing. (check inside kernel, see code above)
		int sectorWidth		= gpu_blk_size();
		int threadsPerBlock = 32; //_GpuProp.maxThreadsPerBlock;
		int blocksPerGrid   = (((inplane->lengthPx*inplane->lineLen)/sectorWidth) + threadsPerBlock - 1) / threadsPerBlock;

		_screen_fms_sml_kernel <<<blocksPerGrid, threadsPerBlock, 0, pstream->stream >>> (pstream->in, pstream->out, pstream->ta, pstream->df, inplane->lineLen, inplane->lengthPx, sectorWidth, limitM, limitL);

		hipStreamSynchronize(pstream->stream);
		pstream->time[2]=rx_get_ticks();
	//	TrPrintfL(TRUE, "GPU[%d].process done", threadNo);

		hipMemcpyAsync(outplane->buffer, pstream->out, outplane->lengthPx * outplane->lineLen, hipMemcpyDeviceToHost, pstream->stream);

		hipStreamSynchronize(pstream->stream);
		pstream->time[3]=rx_get_ticks();
	//	TrPrintfL(TRUE, "GPU[%d].memcpy(out) done", threadNo);

		pstream->Time[0] = pstream->time[1]-pstream->time[0];
		pstream->Time[1] = pstream->time[2]-pstream->time[1];
		pstream->Time[2] = pstream->time[3]-pstream->time[2];
	}
	else
	{
		int time0=rx_get_ticks();
		for (UINT32 y=0; y<inplane->lengthPx; y++)
		{
			_screen_fms_sml(y*inplane->lineLen, inplane->buffer, outplane->buffer, pplaneScreenConfig->TA->ta16, pplaneScreenConfig->densityFactor, inplane->lineLen, inplane->lengthPx, inplane->lineLen, limitM, limitL);
		}
		_Time[0]=0;
		_Time[1]=rx_get_ticks()-time0;
		_Time[2]=0;
	}
	return REPLY_OK;
}

//--- gpu_time ----------------------
int  gpu_time(int no)
{
	if (_GPU_Present)
	{
		return (int)(_GPU_Stream[0].Time[no]+0.5);
	}
	return (int)(_Time[no]+0.5);
}

#ifdef __cplusplus
}
#endif